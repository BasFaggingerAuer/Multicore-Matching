#include "hip/hip_runtime.h"
/*
Copyright 2011, Bas Fagginger Auer.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <iostream>
#include <exception>
#include <string>
#include <algorithm>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <hip/device_functions.h>
//CUDA 3.2 does not seem to make definitions for texture types.
#ifndef hipTextureType1D
#define hipTextureType1D 0x01
#endif

#include "matchgpu.h"

using namespace std;
using namespace mtc;

__constant__ uint dSelectBarrier = 0x8000000;

GraphMatchingGPU::GraphMatchingGPU(const Graph &_graph, const int &_threadsPerBlock, const unsigned int &_selectBarrier) :
		threadsPerBlock(_threadsPerBlock),
		GraphMatching(_graph),
		selectBarrier(_selectBarrier)
{
	//Allocate memory to store the graph on the device.
	if (hipMalloc(&dneighbourRanges, sizeof(int2)*graph.neighbourRanges.size()) != hipSuccess
		|| hipMalloc(&dneighbours, sizeof(int)*graph.neighbours.size()) != hipSuccess)
	{
		cerr << "Not enough memory on device to store this graph!" << endl;
		throw exception();
	}

	//Copy graph data to device.
	if (hipMemcpy(dneighbourRanges, &graph.neighbourRanges[0], sizeof(int2)*graph.neighbourRanges.size(), hipMemcpyHostToDevice) != hipSuccess
		|| hipMemcpy(dneighbours, &graph.neighbours[0], sizeof(int)*graph.neighbours.size(), hipMemcpyHostToDevice) != hipSuccess)
	{
		cerr << "Unable to transfer graph data to device!" << endl;
		throw exception();
	}
	/* This doesn't work on a K40m with cuda 11.  use cuda 10! 
		Kepler GPU's are deprecated in CUDA 11.
		https://arnon.dk/tag/nvcc-flags/
	*/
	//Set select barrier.
	if (hipMemcpyToSymbol(HIP_SYMBOL(dSelectBarrier), &selectBarrier, sizeof(uint)) != hipSuccess)
	{
		cerr << "Unable to set selection barrier!" << endl;
		throw exception();
	}
}

GraphMatchingGPU::~GraphMatchingGPU()
{
	//Free all graph data on the GPU.
	hipFree(dneighbours);
	hipFree(dneighbourRanges);
}

GraphMatchingGPURandom::GraphMatchingGPURandom(const Graph &_graph, const int &_nrThreads, const unsigned int &_selectBarrier) :
		GraphMatchingGPU(_graph, _nrThreads, _selectBarrier)
{

}

GraphMatchingGPURandom::~GraphMatchingGPURandom()
{

}

GraphMatchingGPURandomMaximal::GraphMatchingGPURandomMaximal(const Graph &_graph, const int &_nrThreads, const unsigned int &_selectBarrier) :
		GraphMatchingGPU(_graph, _nrThreads, _selectBarrier)
{

}

GraphMatchingGPURandomMaximal::~GraphMatchingGPURandomMaximal()
{

}

GraphMatchingGPUWeighted::GraphMatchingGPUWeighted(const Graph &_graph, const int &_nrThreads, const unsigned int &_selectBarrier) :
		GraphMatchingGPU(_graph, _nrThreads, _selectBarrier)
{
	assert(graph.neighbourWeights.size() == graph.neighbours.size());

	//Allocate memory on the device to store the graph weights.
	if (hipMalloc(&dweights, sizeof(float)*graph.neighbourWeights.size()) != hipSuccess)
	{
		cerr << "Not enough memory on device to store graph weights!" << endl;
		throw exception();
	}

	//Copy weights.
	if (hipMemcpy(dweights, &graph.neighbourWeights[0], sizeof(float)*graph.neighbourWeights.size(), hipMemcpyHostToDevice) != hipSuccess)
	{
		cerr << "Unable to transfer graph weights to device!" << endl;
		throw exception();
	}
}

GraphMatchingGPUWeighted::~GraphMatchingGPUWeighted()
{
	//Free weights.
	hipFree(dweights);
}

GraphMatchingGPUWeightedMaximal::GraphMatchingGPUWeightedMaximal(const Graph &_graph, const int &_nrThreads, const unsigned int &_selectBarrier) :
		GraphMatchingGPU(_graph, _nrThreads, _selectBarrier)
{
	assert(graph.neighbourWeights.size() == graph.neighbours.size());

	//Allocate memory on the device to store the graph weights.
	if (hipMalloc(&dweights, sizeof(float)*graph.neighbourWeights.size()) != hipSuccess)
	{
		cerr << "Not enough memory on device to store graph weights!" << endl;
		throw exception();
	}

	//Copy weights.
	if (hipMemcpy(dweights, &graph.neighbourWeights[0], sizeof(float)*graph.neighbourWeights.size(), hipMemcpyHostToDevice) != hipSuccess)
	{
		cerr << "Unable to transfer graph weights to device!" << endl;
		throw exception();
	}
}

GraphMatchingGPUWeightedMaximal::~GraphMatchingGPUWeightedMaximal()
{
	//Free weights.
	hipFree(dweights);
}

GraphMatchingGeneralGPURandom::GraphMatchingGeneralGPURandom(const Graph &_graph, const int &_nrThreads, const unsigned int &_selectBarrier) :
		GraphMatchingGPU(_graph, _nrThreads, _selectBarrier)
{

}

GraphMatchingGeneralGPURandom::~GraphMatchingGeneralGPURandom()
{

}

//==== Kernel variables ====
__device__ int dkeepMatching;

texture<int2, hipTextureType1D, hipReadModeElementType> neighbourRangesTexture;
texture<int, hipTextureType1D, hipReadModeElementType> neighboursTexture;
texture<float, hipTextureType1D, hipReadModeElementType> weightsTexture;

//==== General matching kernels ====
/*
   Match values match[i] have the following interpretation for a vertex i:
   0   = blue,
   1   = red,
   2   = unmatchable (all neighbours of i have been matched),
   3   = reserved,
   >=4 = matched.
*/

//Nothing-up-my-sleeve working constants from SHA-256.
__constant__ const uint dMD5K[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
				0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
				0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
				0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
				0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
				0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
				0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
				0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

//Rotations from MD5.
__constant__ const uint dMD5R[64] = {7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
				5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20,
				4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
				6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21};

#define LEFTROTATE(a, b) (((a) << (b)) | ((a) >> (32 - (b))))

__global__ void gSelect(int *match, const int nrVertices, const uint random)
{
	//Determine blue and red groups using MD5 hashing.
	//Based on the Wikipedia MD5 hashing pseudocode (http://en.wikipedia.org/wiki/MD5).
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;

	//Can this vertex still be matched?
	if (match[i] >= 2) return;

	//Start hashing.
	uint h0 = 0x67452301, h1 = 0xefcdab89, h2 = 0x98badcfe, h3 = 0x10325476;
	uint a = h0, b = h1, c = h2, d = h3, e, f, g = i;

	for (int j = 0; j < 16; ++j)
	{
		f = (b & c) | ((~b) & d);

		e = d;
		d = c;
		c = b;
		b += LEFTROTATE(a + f + dMD5K[j] + g, dMD5R[j]);
		a = e;

		h0 += a;
		h1 += b;
		h2 += c;
		h3 += d;

		g *= random;
	}
	
	match[i] = ((h0 + h1 + h2 + h3) < dSelectBarrier ? 0 : 1);
}

__global__ void gSelect(int *match, int *sense, int *heads, int *tails, const int nrVertices, const uint random)
{
	//Determine blue and red groups using MD5 hashing.
	//Based on the Wikipedia MD5 hashing pseudocode (http://en.wikipedia.org/wiki/MD5).
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices || match[i] >= 2) return;

	// Is this vertex a head or a tail? Else decolor
	uint tail = tails[i];
	uint head = heads[i];
	bool singleton = (head == tail);
	//if (threadIdx.x == 0)
	//printf("vertex %d head %d, tail %d\n", i, head, tail);

	if ( head != i && tail != i) match[i] = 2;

	//Can this vertex still be matched?
	if (match[i] >= 2) return;

	//Start hashing.
	uint h0 = 0x67452301, h1 = 0xefcdab89, h2 = 0x98badcfe, h3 = 0x10325476;
	uint a = h0, b = h1, c = h2, d = h3, e, f;

	// match heads and tails same match by using min as g.
	// Hash color of set
	uint g = min(tail, head);

	for (int j = 0; j < 16; ++j)
	{
		f = (b & c) | ((~b) & d);

		e = d;
		d = c;
		c = b;
		b += LEFTROTATE(a + f + dMD5K[j] + g, dMD5R[j]);
		a = e;

		h0 += a;
		h1 += b;
		h2 += c;
		h3 += d;

		g *= random;
	}
	
	uint color = ((h0 + h1 + h2 + h3) < dSelectBarrier ? 0 : 1);
	match[i] = color;
	// Singletons are made the right sense for their color to promote matching.
	// Red(-) and Blue(+)
	if (singleton){
		sense[i] = color;
	}
	else
	{
		// Currently sense is rehashed every iteration
		// to prevent this use color==2 to prevent
		// replace else with 
		// else if (color != 2)
		// Hash sense
		uint g = max(tail, head);
		bool mask = (g == i);

		for (int j = 0; j < 16; ++j)
		{
			f = (b & c) | ((~b) & d);

			e = d;
			d = c;
			c = b;
			b += LEFTROTATE(a + f + dMD5K[j] + g, dMD5R[j]);
			a = e;

			h0 += a;
			h1 += b;
			h2 += c;
			h3 += d;

			g *= random;
		}
		// Notice how in each case i and j have opposite senses.
		// Truth Table to Check //
		//                   
		//     a    b    a^b
		//C1
		//i // 0    0    0   
		//j // 0    1    1 
		//C3
		//i // 0    1    1   
		//j // 0    0    0  
		//C3
		//i // 1    0    1   
		//j // 1    1    0 
		//C4
		//i // 1    1    0   
		//j // 1    0    1  
		bool a = (bool)((h0 + h1 + h2 + h3) < dSelectBarrier ? 0 : 1);
		bool b = mask;
		//bool XOR(bool a, bool b)
		sense[i] = (a + b) % 2;
	}
	//if (threadIdx.x == 0)
	//printf("vert %d, color %d, sense %d\n", i, color, sense[i]);
}

__global__ void gaSelect(int *match, const int nrVertices, const uint random)
{
	//Determine blue and red groups using MD5 hashing.
	//Based on the Wikipedia MD5 hashing pseudocode (http://en.wikipedia.org/wiki/MD5).
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;

	//Can this vertex still be matched?
	if (match[i] >= 2) return;

	//Use atomic operations to indicate whether we are done.
	//atomicCAS(&dkeepMatching, 0, 1);
	dkeepMatching = 1;

	//Start hashing.
	uint h0 = 0x67452301, h1 = 0xefcdab89, h2 = 0x98badcfe, h3 = 0x10325476;
	uint a = h0, b = h1, c = h2, d = h3, e, f, g = i;

	for (int j = 0; j < 16; ++j)
	{
		f = (b & c) | ((~b) & d);

		e = d;
		d = c;
		c = b;
		b += LEFTROTATE(a + f + dMD5K[j] + g, dMD5R[j]);
		a = e;

		h0 += a;
		h1 += b;
		h2 += c;
		h3 += d;

		g *= random;
	}
	
	match[i] = ((h0 + h1 + h2 + h3) < dSelectBarrier ? 0 : 1);
}

__global__ void gMatch(int *match, const int *requests, const int nrVertices)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;

	const int r = requests[i];

	//Only unmatched vertices make requests.
	if (r == nrVertices + 1)
	{
		//This is vertex without any available neighbours, discard it.
		match[i] = 2;
	}
	else if (r < nrVertices)
	{
		//This vertex has made a valid request.
		if (requests[r] == i)
		{
			//Match the vertices if the request was mutual.
			match[i] = 4 + min(i, r);
		}
	}
}


__global__ void gMatch(int *match, int *sense, int *heads, int *tails, int *flinkedlist, int *blinkedlist, const int *requests, const int nrVertices)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;

	const int r = requests[i];

	// Only unmatched vertices make requests.
	// Need to reset this every coarsening iteration for head and tails?
	if (r == nrVertices + 1)
	{
		// This is vertex Blue(+) without any Blue or Red neighbors
		// Discard it and flip sense.
		match[i] = 2;
	}
	// Only true if a B+ is neighbors with a R- 
	// The pairing might have not occurred because of competition.
	else if (r < nrVertices)
	{
		// This vertex has made a valid request.
		// Match the vertices if the request was mutual.
		// R+ paired with a B-  -> R+.R- or B+.B-
		// R+.R- paired with a B+.B-  -> R+.x.x.R- or B+.x.x.B-
		if (requests[r] == i){
			// No race-conditions:
			// + sense: read heads[r] and write into heads[i], tails[i] unchanged.
			// - sense: read tails[r] and write into tails[i], heads[i] unchanged.

			if(sense[i]){ 
				// Negative sense, update head
				tails[i] = tails[r];
				printf("tail %d\n", tails[i]);
				// heads[i] isn't thread-sensitive since I am the (-) end
				match[tails[i]] = 4 + min(heads[i], tails[i]);
			} else {
				// Positive sense, update tail
				// Update head
				heads[i] = heads[r];
				// tails[i] isn't thread-sensitive since I am the (+) end
				match[heads[i]] = 4 + min(heads[i], tails[i]);
			}
			flinkedlist[i] = r;
			blinkedlist[r] = i;
			//printf("SUCCESS\n");
		}
	}
}
/**
Precondition: Graph is composed of colored heads and tails with 
dead internal path nodes.  Also, there are entirely dead nodes/paths.
Postcondition: Graph is paritioned into sets with unique colors.		
Requirement: Matching is completed. Calling this while matching 
will produce incorrect results.
Usage: Primarily for visualization purposes.
*/
__global__ void gUncoarsen(int *match, int *heads, int *tails, int *flinkedlist, int *blinkedlist, const int nrVertices)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;
	uint head = heads[i];
	// Only color from heads to prevent unneccessary work.
	if (head != i) return;
	uint tail = tails[i];
	int color = match[i];
	// Entirely dead paths need to be revived.
	// Also unmatched red/blues need to given a unique color.
	if (color < 4){
		color = i;
		match[i] = color;
	}
	uint next = flinkedlist[i];
	uint prev = blinkedlist[i];
	bool directionToTail;
	// Equivalent to checking if head=tail
	if (next == prev)
		return;
	else if (next == i)
		directionToTail = 0;
	else
		directionToTail = 1;

	do {
		match[i] = color;
		i = (directionToTail ? flinkedlist[i] : blinkedlist[i]);
	} while (tail != i);
	// Color tail
	match[i] = color;
}

//==== Random greedy matching kernels ====
__global__ void grRequest(int *requests, const int *match, const int nrVertices)
{
	//Let all blue vertices make requests.
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;
	
	const int2 indices = tex1Dfetch(neighbourRangesTexture, i);

	//Look at all blue vertices and let them make requests.
	if (match[i] == 0)
	{
		int dead = 1;

		for (int j = indices.x; j < indices.y; ++j)
		{
			const int ni = tex1Dfetch(neighboursTexture, j);
			const int nm = match[ni];

			//Do we have an unmatched neighbour?
			if (nm < 4)
			{
				//Is this neighbour red?
				if (nm == 1)
				{
					//Propose to this neighbour.
					requests[i] = ni;
					return;
				}
				
				dead = 0;
			}
		}
		requests[i] = nrVertices + dead;
	}
	else
	{
		// If I'm red
		//Clear request value.
		requests[i] = nrVertices;
	}
}


//==== Random greedy matching kernels ====
__global__ void grRequest(int *requests, const int *match, const int *sense, const int *forwardlinkedlist, const int *backwardlinkedlist, const int nrVertices)
{
	//Let all blue (+) vertices make requests.
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;
	
	const int2 indices = tex1Dfetch(neighbourRangesTexture, i);

	//Look at all blue (+) vertices and let them make requests.
	if (match[i] == 0 && sense[i] == 0)
	{
		int noUnmatchedNeighborExists = 1;
		// One of these must be myself and the other might be myself (singleton)
		// Since I allow quick sense flipping, it is unclear whether head->me or tail->me
		// All that is known is I am either head or tail and at most one one my neighbors
		// can be in my matching.  Therefore, just check each neighbor against both directions.
		const int nf = forwardlinkedlist[i];
		const int nb = backwardlinkedlist[i];		
		for (int j = indices.x; j < indices.y; ++j)
		{
			const int ni = tex1Dfetch(neighboursTexture, j);
			// Prevents matching an already matched neighbor
			// We would never successfully rematch
			// but the "noUnmatchedNeighborExists" 
			// flag will never be set for pairs
			// without this continue statement.
			// r+.-r-, b+.b-; there is a colored neighbor.
			if (nf == ni || nb == ni) continue;
			const int nm = match[ni];
			//Do we have an unmatched neighbour?
			// 0 : Blue; 1 : Red, 2 
			// Blue or Red
			if (nm < 4)
			{
				// Negative sense 
				if (sense[ni] == 1){
					//Is this neighbour red?
					if (nm == 1)
					{
						//Propose to this red(-) neighbour.
						requests[i] = ni;
						//printf("I %d requested %d\n", i, ni);
						return;
					}
				}
				// Neighbor is : [red(+) or blue(-)]
				noUnmatchedNeighborExists = 0;
			}
		}
		// N   -> Neighbors : [red(+), blue(-)] -> recolor
		// N+1 -> No unmatched neighbors -> decolor
		requests[i] = nrVertices + noUnmatchedNeighborExists;
	}
	else
	{
		// If I'm red or blue (-)
		//Clear request value.
		requests[i] = nrVertices;
	}
}

__global__ void grRespond(int *requests, const int *match, const int nrVertices)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;
	
	const int2 indices = tex1Dfetch(neighbourRangesTexture, i);

	//Look at all red vertices.
	if (match[i] == 1)
	{
		//Select first available proposer.
		for (int j = indices.x; j < indices.y; ++j)
		{
			const int ni = tex1Dfetch(neighboursTexture, j);

			//Only respond to blue neighbours.
			if (match[ni] == 0)
			{
				//Avoid data thrashing be only looking at the request value of blue neighbours.
				if (requests[ni] == i)
				{
					requests[i] = ni;
					return;
				}
			}
		}
	}
}


__global__ void grRespond(int *requests, const int *match, const int *sense, const int nrVertices)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;
	
	const int2 indices = tex1Dfetch(neighbourRangesTexture, i);

	//Look at all red (-) vertices.
	if (match[i] == 1 && sense[i] == 1)
	{
		//Select first available proposer.
		for (int j = indices.x; j < indices.y; ++j)
		{
			const int ni = tex1Dfetch(neighboursTexture, j);
			// Dont have to worry about evaluating already matched vertices
			// Since these must be opposite color and sense.
			//Only respond to blue (+) neighbours.
			if (match[ni] == 0 && sense[ni] == 0)
			{
				//Avoid data thrashing be only looking at the request value of blue neighbours.
				if (requests[ni] == i)
				{
					requests[i] = ni;
					//printf("I %d responded to %d\n", i, ni);
					return;
				}
			}
		}
	}
}

//==== Weighted greedy matching kernels ====
__global__ void gwRequest(int *requests, const int *match, const int nrVertices)
{
	//Let all blue vertices make requests.
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;
	
	const int2 indices = tex1Dfetch(neighbourRangesTexture, i);

	//Look at all blue vertices and let them make requests.
	if (match[i] == 0)
	{
		float maxWeight = -1.0;
		int candidate = nrVertices;
		int dead = 1;

		for (int j = indices.x; j < indices.y; ++j)
		{
			//Only propose to red neighbours.
			const int ni = tex1Dfetch(neighboursTexture, j);
			const int nm = match[ni];

			//Do we have an unmatched neighbour?
			if (nm < 4)
			{
				//Is this neighbour red?
				if (nm == 1)
				{
					//Propose to the heaviest neighbour.
					const float nw = tex1Dfetch(weightsTexture, j);

					if (nw > maxWeight)
					{
						maxWeight = nw;
						candidate = ni;
					}
				}
				
				dead = 0;
			}
		}

		requests[i] = candidate + dead;
	}
	else
	{
		//Clear request value.
		requests[i] = nrVertices;
	}
}

__global__ void gwRespond(int *requests, const int *match, const int nrVertices)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= nrVertices) return;
	
	const int2 indices = tex1Dfetch(neighbourRangesTexture, i);

	//Look at all red vertices.
	if (match[i] == 1)
	{
		float maxWeight = -1;
		int candidate = nrVertices;

		//Select heaviest available proposer.
		for (int j = indices.x; j < indices.y; ++j)
		{
			const int ni = tex1Dfetch(neighboursTexture, j);

			//Only respond to blue neighbours.
			if (match[ni] == 0)
			{
				if (requests[ni] == i)
				{
					const float nw = tex1Dfetch(weightsTexture, j);

					if (nw > maxWeight)
					{
						maxWeight = nw;
						candidate = ni;
					}
				}
			}
		}

		if (candidate < nrVertices)
		{
			requests[i] = candidate;
		}
	}
}

void GraphMatchingGPURandom::performMatching(vector<int> &match, hipEvent_t &t1, hipEvent_t &t2, vector<int> & h, vector<int> & t, vector<int> & fll) const
{
	//Creates a greedy random matching on the GPU.
	//Assumes the current matching is empty.

	assert((int)match.size() == graph.nrVertices);
	
	//Setup textures.
	hipChannelFormatDesc neighbourRangesTextureDesc = hipCreateChannelDesc<int2>();

	neighbourRangesTexture.addressMode[0] = hipAddressModeWrap;
	neighbourRangesTexture.filterMode = hipFilterModePoint;
	neighbourRangesTexture.normalized = false;
	hipBindTexture(0, neighbourRangesTexture, (void *)dneighbourRanges, neighbourRangesTextureDesc, sizeof(int2)*graph.neighbourRanges.size());
	
	hipChannelFormatDesc neighboursTextureDesc = hipCreateChannelDesc<int>();

	neighboursTexture.addressMode[0] = hipAddressModeWrap;
	neighboursTexture.filterMode = hipFilterModePoint;
	neighboursTexture.normalized = false;
	hipBindTexture(0, neighboursTexture, (void *)dneighbours, neighboursTextureDesc, sizeof(int)*graph.neighbours.size());

	//Allocate necessary buffers on the device.
	int *dmatch, *drequests;

	if (hipMalloc(&dmatch, sizeof(int)*graph.nrVertices) != hipSuccess
		|| hipMalloc(&drequests, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Not enough memory on device!" << endl;
		throw exception();
	}

	//Clear matching.
	if (hipMemset(dmatch, 0, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Unable to clear matching on device!" << endl;
		throw exception();
	}

	//Perform matching.
	int blocksPerGrid = (graph.nrVertices + threadsPerBlock - 1)/threadsPerBlock;
	
	//Perform all stages, one-by-one.
#ifndef NDEBUG
	hipGetLastError();
#endif

	hipEventRecord(t1, 0);
	hipEventSynchronize(t1);

#ifdef MATCH_INTERMEDIATE_COUNT
	cout << "0\t0\t0" << endl;
#endif

	for (int i = 0; i < NR_MATCH_ROUNDS; ++i)
	{
		gSelect<<<blocksPerGrid, threadsPerBlock>>>(dmatch, graph.nrVertices, rand());
		grRequest<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, graph.nrVertices);
		grRespond<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, graph.nrVertices);
		gMatch<<<blocksPerGrid, threadsPerBlock>>>(dmatch, drequests, graph.nrVertices);

#ifdef MATCH_INTERMEDIATE_COUNT
		hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost);
		
		double weight = 0;
		long size = 0;

		getWeight(weight, size, match, graph);

		cout << i + 1 << "\t" << weight << "\t" << size << endl;
#endif
	}
	
	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);

#ifndef NDEBUG
	hipError_t error;

	if ((error = hipGetLastError()) != hipSuccess)
	{
		cerr << "A CUDA error occurred during the matching process: " << hipGetErrorString(error) << endl;
		throw exception();
	}
#endif

	//Copy obtained matching on the device back to the host.
	if (hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost) != hipSuccess)
	{
		cerr << "Unable to retrieve data!" << endl;
		throw exception();
	}

	//Free memory.
	hipFree(drequests);
	hipFree(dmatch);
	hipUnbindTexture(neighboursTexture);
	hipUnbindTexture(neighbourRangesTexture);
}

void GraphMatchingGeneralGPURandom::performMatching(vector<int> &match, hipEvent_t &t1, hipEvent_t &t2, vector<int> & h, vector<int> & t, vector<int> & fll) const
{
	//Creates a greedy random matching on the GPU.
	//Assumes the current matching is empty.
	std::cout << "GraphMatchingGeneralGPURandom" << std::endl;

	assert((int)match.size() == graph.nrVertices);
	
	//Setup textures.
	hipChannelFormatDesc neighbourRangesTextureDesc = hipCreateChannelDesc<int2>();

	neighbourRangesTexture.addressMode[0] = hipAddressModeWrap;
	neighbourRangesTexture.filterMode = hipFilterModePoint;
	neighbourRangesTexture.normalized = false;
	hipBindTexture(0, neighbourRangesTexture, (void *)dneighbourRanges, neighbourRangesTextureDesc, sizeof(int2)*graph.neighbourRanges.size());
	
	hipChannelFormatDesc neighboursTextureDesc = hipCreateChannelDesc<int>();

	neighboursTexture.addressMode[0] = hipAddressModeWrap;
	neighboursTexture.filterMode = hipFilterModePoint;
	neighboursTexture.normalized = false;
	hipBindTexture(0, neighboursTexture, (void *)dneighbours, neighboursTextureDesc, sizeof(int)*graph.neighbours.size());

	//Allocate necessary buffers on the device.
	// dlinkedlists - to generalize matching to n edges
	// dtails - to quickly flip sense of strand
	// dmatch - same as singleton implementation
	// dsense - indicates directionality of strand
	int *dforwardlinkedlist, *dbackwardlinkedlist, *dmatch, *drequests, *dsense, *dheads, *dtails;

	if (hipMalloc(&drequests, sizeof(int)*graph.nrVertices) != hipSuccess ||  
		hipMalloc(&dmatch, sizeof(int)*graph.nrVertices) != hipSuccess || 
		hipMalloc(&dsense, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Not enough memory on device!" << endl;
		throw exception();
	}

	thrust::device_vector<int>H(graph.nrVertices);
	thrust::sequence(H.begin(),H.end());
	dheads = thrust::raw_pointer_cast(&H[0]);
	
	thrust::device_vector<int>T(graph.nrVertices);
	thrust::sequence(T.begin(),T.end());
	dtails = thrust::raw_pointer_cast(&T[0]);

	thrust::device_vector<int>dfll(graph.nrVertices);
	thrust::sequence(dfll.begin(),dfll.end());
	dforwardlinkedlist = thrust::raw_pointer_cast(&dfll[0]);
	
	thrust::device_vector<int>bll(graph.nrVertices);
	thrust::sequence(bll.begin(),bll.end());
	dbackwardlinkedlist = thrust::raw_pointer_cast(&bll[0]);

	//Clear matching.
	if (hipMemset(dforwardlinkedlist, 0, sizeof(int)*graph.nrVertices) != hipSuccess ||
		hipMemset(dbackwardlinkedlist, 0, sizeof(int)*graph.nrVertices))
	{
		cerr << "Unable to clear matching on device!" << endl;
		throw exception();
	}

	//Perform matching.
	int blocksPerGrid = (graph.nrVertices + threadsPerBlock - 1)/threadsPerBlock;
	
	//Perform all stages, one-by-one.
#ifndef NDEBUG
	hipGetLastError();
#endif

	hipEventRecord(t1, 0);
	hipEventSynchronize(t1);

#ifdef MATCH_INTERMEDIATE_COUNT
	cout << "0\t0\t0" << endl;
#endif
	int maxlength = 3;
	for (int lengthOfPath = 0; lengthOfPath < maxlength; ++lengthOfPath){
		// The inner loop methods generalize from singletons to linked lists of any length
		// Therefore, all we need to do is reset the match repeat the inner loop.
		// Each inner loop call adds at most one edge to a path.
		// However, after the first inner loop call, which is guarunteed
		// to match at least half the graph, success is random.
		if (hipMemset(dmatch, 0, sizeof(int)*graph.nrVertices) != hipSuccess)
		{
			cerr << "Unable to clear matching on device!" << endl;
			throw exception();
		}

		for (int i = 0; i < NR_MATCH_ROUNDS; ++i)
		{
			gSelect<<<blocksPerGrid, threadsPerBlock>>>(dmatch, dsense, dheads, dtails, graph.nrVertices, rand());
			grRequest<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, dsense, dforwardlinkedlist, dbackwardlinkedlist, graph.nrVertices);
			grRespond<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, dsense, graph.nrVertices);
			gMatch<<<blocksPerGrid, threadsPerBlock>>>(dmatch, dsense, dheads, dtails, 
														dforwardlinkedlist, dbackwardlinkedlist, 
														drequests, graph.nrVertices);

	#ifdef MATCH_INTERMEDIATE_COUNT
			hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost);
			
			double weight = 0;
			long size = 0;

			getWeight(weight, size, match, graph);

			cout << i + 1 << "\t" << weight << "\t" << size << endl;
	#endif
		}
		
		hipEventRecord(t2, 0);
		hipEventSynchronize(t2);
	}
#ifndef NDEBUG
	hipError_t error;

	if ((error = hipGetLastError()) != hipSuccess)
	{
		cerr << "A CUDA error occurred during the matching process: " << hipGetErrorString(error) << endl;
		throw exception();
	}
#endif


	// call uncoarsen for viz
	//#ifdef UNCOARSEN_GRAPH	
	gUncoarsen<<<blocksPerGrid, threadsPerBlock>>>(dmatch, dheads, dtails, 
													dforwardlinkedlist, dbackwardlinkedlist, 
													graph.nrVertices);
	//#endif

	//Copy obtained matching on the device back to the host.
	if (hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost) != hipSuccess)
	{
		cerr << "Unable to retrieve data!" << endl;
		throw exception();
	}

	thrust::host_vector<int>htheads;
	thrust::host_vector<int>httails;
	thrust::host_vector<int>htforwardlinkedlist;

	htheads = H;
	httails = T;
	htforwardlinkedlist = fll;

	for (for int i = 0; i < httails.size(); ++i)
		std::cout << "tail " << httails[i] << std::endl;


	thrust::copy(htheads.begin(), htheads.end(), h.begin());
	thrust::copy(httails.begin(), httails.end(), t.begin());
	thrust::copy(htforwardlinkedlist.begin(), htforwardlinkedlist.end(), fll.begin());
	std::cout << "finishd copying" << std::endl;
	//Free memory.
	hipFree(drequests);
	hipFree(dmatch);
	hipFree(dsense);
	hipUnbindTexture(neighboursTexture);
	hipUnbindTexture(neighbourRangesTexture);
}

void GraphMatchingGPURandomMaximal::performMatching(vector<int> &match, hipEvent_t &t1, hipEvent_t &t2, vector<int> & h, vector<int> & t, vector<int> & fll) const
{
	//Creates a greedy random maximal matching on the GPU using atomic operations.
	//Assumes the current matching is empty.

	assert((int)match.size() == graph.nrVertices);
	
	//Setup textures.
	hipChannelFormatDesc neighbourRangesTextureDesc = hipCreateChannelDesc<int2>();

	neighbourRangesTexture.addressMode[0] = hipAddressModeWrap;
	neighbourRangesTexture.filterMode = hipFilterModePoint;
	neighbourRangesTexture.normalized = false;
	hipBindTexture(0, neighbourRangesTexture, (void *)dneighbourRanges, neighbourRangesTextureDesc, sizeof(int2)*graph.neighbourRanges.size());
	
	hipChannelFormatDesc neighboursTextureDesc = hipCreateChannelDesc<int>();

	neighboursTexture.addressMode[0] = hipAddressModeWrap;
	neighboursTexture.filterMode = hipFilterModePoint;
	neighboursTexture.normalized = false;
	hipBindTexture(0, neighboursTexture, (void *)dneighbours, neighboursTextureDesc, sizeof(int)*graph.neighbours.size());

	//Allocate necessary buffers on the device.
	int *dmatch, *drequests;

	if (hipMalloc(&dmatch, sizeof(int)*graph.nrVertices) != hipSuccess
		|| hipMalloc(&drequests, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Not enough memory on device!" << endl;
		throw exception();
	}

	//Clear matching.
	if (hipMemset(dmatch, 0, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Unable to clear matching on device!" << endl;
		throw exception();
	}

	//Perform matching.
	int keepMatching = 1, count = 0;
	int blocksPerGrid = (graph.nrVertices + threadsPerBlock - 1)/threadsPerBlock;

	//Perform all stages, one-by-one.
#ifndef NDEBUG
	hipGetLastError();
#endif

	hipEventRecord(t1, 0);
	hipEventSynchronize(t1);

	while (keepMatching == 1 && ++count < NR_MAX_MATCH_ROUNDS)
	{
		keepMatching = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(dkeepMatching), &keepMatching, sizeof(int));

		gaSelect<<<blocksPerGrid, threadsPerBlock>>>(dmatch, graph.nrVertices, rand());
		grRequest<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, graph.nrVertices);
		grRespond<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, graph.nrVertices);
		gMatch<<<blocksPerGrid, threadsPerBlock>>>(dmatch, drequests, graph.nrVertices);

		hipMemcpyFromSymbol(&keepMatching, HIP_SYMBOL(dkeepMatching), sizeof(int));
	}
	
	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);

#ifndef NDEBUG
	hipError_t error;

	if ((error = hipGetLastError()) != hipSuccess)
	{
		cerr << "A CUDA error occurred during the matching process: " << hipGetErrorString(error) << endl;
		throw exception();
	}
#endif

	//Copy obtained matching on the device back to the host.
	if (hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost) != hipSuccess)
	{
		cerr << "Unable to retrieve data!" << endl;
		throw exception();
	}

	//Free memory.
	hipFree(drequests);
	hipFree(dmatch);
	hipUnbindTexture(neighboursTexture);
	hipUnbindTexture(neighbourRangesTexture);
}

void GraphMatchingGPUWeighted::performMatching(vector<int> &match, hipEvent_t &t1, hipEvent_t &t2, vector<int> & h, vector<int> & t, vector<int> & fll) const
{
	//Creates a greedy weighted matching on the GPU.
	//Assumes the current matching is empty.

	assert((int)match.size() == graph.nrVertices);
	
	//Setup textures.
	hipChannelFormatDesc neighbourRangesTextureDesc = hipCreateChannelDesc<int2>();

	neighbourRangesTexture.addressMode[0] = hipAddressModeWrap;
	neighbourRangesTexture.filterMode = hipFilterModePoint;
	neighbourRangesTexture.normalized = false;
	hipBindTexture(0, neighbourRangesTexture, (void *)dneighbourRanges, neighbourRangesTextureDesc, sizeof(int2)*graph.neighbourRanges.size());
	
	hipChannelFormatDesc neighboursTextureDesc = hipCreateChannelDesc<int>();

	neighboursTexture.addressMode[0] = hipAddressModeWrap;
	neighboursTexture.filterMode = hipFilterModePoint;
	neighboursTexture.normalized = false;
	hipBindTexture(0, neighboursTexture, (void *)dneighbours, neighboursTextureDesc, sizeof(int)*graph.neighbours.size());

	hipChannelFormatDesc weightsTextureDesc = hipCreateChannelDesc<float>();

	weightsTexture.addressMode[0] = hipAddressModeWrap;
	weightsTexture.filterMode = hipFilterModePoint;
	weightsTexture.normalized = false;
	hipBindTexture(0, weightsTexture, (void *)dweights, weightsTextureDesc, sizeof(float)*graph.neighbourWeights.size());

	//Allocate necessary buffers on the device.
	int *dmatch, *drequests;

	if (hipMalloc(&dmatch, sizeof(int)*graph.nrVertices) != hipSuccess
		|| hipMalloc(&drequests, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Not enough memory on device!" << endl;
		throw exception();
	}

	//Clear matching.
	if (hipMemset(dmatch, 0, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Unable to clear matching on device!" << endl;
		throw exception();
	}

	//Perform matching.
	int blocksPerGrid = (graph.nrVertices + threadsPerBlock - 1)/threadsPerBlock;

	//Perform all stages, one-by-one.
#ifndef NDEBUG
	hipGetLastError();
#endif

	hipEventRecord(t1, 0);
	hipEventSynchronize(t1);

#ifdef MATCH_INTERMEDIATE_COUNT
	cout << "0\t0\t0" << endl;
#endif

	for (int i = 0; i < NR_MATCH_ROUNDS; ++i)
	{
		gSelect<<<blocksPerGrid, threadsPerBlock>>>(dmatch, graph.nrVertices, rand());
		gwRequest<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, graph.nrVertices);
		gwRespond<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, graph.nrVertices);
		gMatch<<<blocksPerGrid, threadsPerBlock>>>(dmatch, drequests, graph.nrVertices);

#ifdef MATCH_INTERMEDIATE_COUNT
		hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost);
		
		double weight = 0;
		long size = 0;

		getWeight(weight, size, match, graph);

		cout << i + 1 << "\t" << weight << "\t" << size << endl;
#endif
	}
	
	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);

#ifndef NDEBUG
	hipError_t error;

	if ((error = hipGetLastError()) != hipSuccess)
	{
		cerr << "A CUDA error occurred during the matching process: " << hipGetErrorString(error) << endl;
		throw exception();
	}
#endif

	//Copy obtained matching on the device back to the host.
	if (hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost) != hipSuccess)
	{
		cerr << "Unable to retrieve data!" << endl;
		throw exception();
	}

	//Free memory.
	hipFree(drequests);
	hipFree(dmatch);

	hipUnbindTexture(weightsTexture);
	hipUnbindTexture(neighboursTexture);
	hipUnbindTexture(neighbourRangesTexture);
}

void GraphMatchingGPUWeightedMaximal::performMatching(vector<int> &match, hipEvent_t &t1, hipEvent_t &t2, vector<int> & h, vector<int> & t, vector<int> & fll) const
{
	//Creates a greedy weighted matching on the GPU.
	//Assumes the current matching is empty.

	assert((int)match.size() == graph.nrVertices);
	
	//Setup textures.
	hipChannelFormatDesc neighbourRangesTextureDesc = hipCreateChannelDesc<int2>();

	neighbourRangesTexture.addressMode[0] = hipAddressModeWrap;
	neighbourRangesTexture.filterMode = hipFilterModePoint;
	neighbourRangesTexture.normalized = false;
	hipBindTexture(0, neighbourRangesTexture, (void *)dneighbourRanges, neighbourRangesTextureDesc, sizeof(int2)*graph.neighbourRanges.size());
	
	hipChannelFormatDesc neighboursTextureDesc = hipCreateChannelDesc<int>();

	neighboursTexture.addressMode[0] = hipAddressModeWrap;
	neighboursTexture.filterMode = hipFilterModePoint;
	neighboursTexture.normalized = false;
	hipBindTexture(0, neighboursTexture, (void *)dneighbours, neighboursTextureDesc, sizeof(int)*graph.neighbours.size());

	hipChannelFormatDesc weightsTextureDesc = hipCreateChannelDesc<float>();

	weightsTexture.addressMode[0] = hipAddressModeWrap;
	weightsTexture.filterMode = hipFilterModePoint;
	weightsTexture.normalized = false;
	hipBindTexture(0, weightsTexture, (void *)dweights, weightsTextureDesc, sizeof(float)*graph.neighbourWeights.size());

	//Allocate necessary buffers on the device.
	int *dmatch, *drequests;

	if (hipMalloc(&dmatch, sizeof(int)*graph.nrVertices) != hipSuccess
		|| hipMalloc(&drequests, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Not enough memory on device!" << endl;
		throw exception();
	}

	//Clear matching.
	if (hipMemset(dmatch, 0, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Unable to clear matching on device!" << endl;
		throw exception();
	}

	//Perform matching.
	int keepMatching = 1, count = 0;
	int blocksPerGrid = (graph.nrVertices + threadsPerBlock - 1)/threadsPerBlock;

	//Perform all stages, one-by-one.
#ifndef NDEBUG
	hipGetLastError();
#endif

	hipEventRecord(t1, 0);
	hipEventSynchronize(t1);

	while (keepMatching == 1 && ++count < NR_MAX_MATCH_ROUNDS)
	{
		keepMatching = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(dkeepMatching), &keepMatching, sizeof(int));

		gaSelect<<<blocksPerGrid, threadsPerBlock>>>(dmatch, graph.nrVertices, rand());
		gwRequest<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, graph.nrVertices);
		gwRespond<<<blocksPerGrid, threadsPerBlock>>>(drequests, dmatch, graph.nrVertices);
		gMatch<<<blocksPerGrid, threadsPerBlock>>>(dmatch, drequests, graph.nrVertices);

		hipMemcpyFromSymbol(&keepMatching, HIP_SYMBOL(dkeepMatching), sizeof(int));
	}
	
	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);

#ifndef NDEBUG
	hipError_t error;

	if ((error = hipGetLastError()) != hipSuccess)
	{
		cerr << "A CUDA error occurred during the matching process: " << hipGetErrorString(error) << endl;
		throw exception();
	}
#endif

	//Copy obtained matching on the device back to the host.
	if (hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost) != hipSuccess)
	{
		cerr << "Unable to retrieve data!" << endl;
		throw exception();
	}

	//Free memory.
	hipFree(drequests);
	hipFree(dmatch);

	hipUnbindTexture(weightsTexture);
	hipUnbindTexture(neighboursTexture);
	hipUnbindTexture(neighbourRangesTexture);
}

